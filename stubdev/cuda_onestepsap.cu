#include "hip/hip_runtime.h"

#include <iostream>

#include "cuda_matmul.cuh"
#include "cuda_onestepsap.cuh"
#include "cuda_reduce.cuh"
#include <hip/hip_runtime.h>

// ========================================================================
// OneStepSapGPU Kernels and Functions with new data struct
// ========================================================================

// Device function to calculate alpha*A + B = C
// A and B are const inputs, C is mutable.
__device__ void SAXPY(double alpha, const Eigen::Map<Eigen::MatrixXd> A,
                      const Eigen::Map<Eigen::MatrixXd> B,
                      Eigen::Map<Eigen::MatrixXd> C) {
  int thread_idx = threadIdx.x;
  int row = A.rows();
  int col = A.cols();

  int num_strides = (A.rows() + 31) / 32;

  for (int i = 0; i < num_strides; i++) {
    int cur_idx = i * 32 + thread_idx;
    if (cur_idx >= row * col) continue;
    int cur_col = cur_idx / row;
    int cur_row = cur_idx % row;

    if (cur_row < row && cur_col < col) {
      C(cur_row, cur_col) = alpha * A(cur_row, cur_col) + B(cur_row, cur_col);
    }
  }
}

// Device function to calculate alpha*(A*B) = C
// A and B are const inputs, C is mutable.
__device__ void MMultiply(double alpha, const Eigen::Map<Eigen::MatrixXd> A,
                          const Eigen::Map<Eigen::MatrixXd> B,
                          Eigen::Map<Eigen::MatrixXd> C, double* sums) {
  int A_row = A.rows();
  int A_col = A.cols();
  int B_col = B.cols();
  int stride = (A_row + 31) / 32;
  int thread_idx = threadIdx.x;

  for (int k = 0; k < B_col; k++) {
    for (int j = 0; j < A_col; j++) {
      for (int i = 0; i < stride; i++) {
        int row = i * 32 + thread_idx;
        int col = j;
        if (row < A_row) {
          if (j == 0) {
            sums[row] = 0.0;
          }

          sums[row] += A(row, col) * B(col, k);

          if (col == A_col - 1) {
            C(row, k) = alpha * sums[row];
          }
        }
      }
    }
  }
}

// Sets lambda_r = 0.5 * gamma.transpose() * R * gamma by modifying `data`
__device__ void CalcRegularizationCost(SAPGPUData* data) {
  double sum = 0.0;
  for (int i = threadIdx.x; i < data->NumContacts(); i += blockDim.x) {
    sum += 0.5 * data->gamma(i).dot(data->R(i).cwiseProduct(data->gamma(i)));
  }
  sum += __shfl_down_sync(0xFFFFFFFF, sum, 16);
  sum += __shfl_down_sync(0xFFFFFFFF, sum, 8);
  sum += __shfl_down_sync(0xFFFFFFFF, sum, 4);
  sum += __shfl_down_sync(0xFFFFFFFF, sum, 2);
  sum += __shfl_down_sync(0xFFFFFFFF, sum, 1);
  if (threadIdx.x == 0) {
    data->regularizer_cost()(0, 0) = sum;
  }
}

// Kernel function serving as a wrapper
__global__ void CalcMomentumCostKernel(SAPGPUData* data) {
  extern __shared__ double sums[];
  int thread_idx = threadIdx.x;
  int equ_idx = blockIdx.x;
  int num_problems = data->NumProblems();

  if (equ_idx >= num_problems) return;

  // Calculate velocity gain
  SAXPY(-1.0, data->v_star(), data->v_guess(), data->velocity_gain());

  // Calculate momentum gain
  MMultiply(1.0, data->dynamics_matrix(), data->velocity_gain(),
            data->momentum_gain(), sums);

  // Calculate momentum cost
  MMultiply(0.5, data->velocity_gain_transpose(), data->momentum_gain(),
            data->momentum_cost(), sums);
}

__global__ void CalcRegularizerCostKernel(SAPGPUData* data) {
  extern __shared__ double sums[];
  int equ_idx = blockIdx.x;
  int num_problems = data->NumProblems();
  int num_contacts = data->NumContacts();

  if (equ_idx >= num_problems) return;

  // Calculate regularization cost
  CalcRegularizationCost(data);
}

__device__ void CalculateHessian(SAPGPUData* data) {
  int num_stride =
      ((3 * data->NumVelocities() * 3 * data->NumVelocities()) + 31) / 32;
  for (int i = 0; i < num_stride; i++) {
    int cur_idx = i * 32 + threadIdx.x;
    if (cur_idx >= 3 * data->NumVelocities() * 3 * data->NumVelocities())
      return;
    int cur_col = cur_idx / (3 * data->NumVelocities());
    int cur_row = cur_idx % (3 * data->NumVelocities());

    if (cur_row < 3 * data->NumVelocities() &&
        cur_col < 3 * data->NumVelocities()) {
      data->H()(cur_row, cur_col) =
          data->J().col(cur_row).dot(data->G_J().col(cur_col));
    }
  }
}

__global__ void CalcHessianKernel(SAPGPUData* data) {
  extern __shared__ double sums[];
  int equ_idx = blockIdx.x;
  int num_problems = data->NumProblems();
  int num_contacts = data->NumContacts();

  if (equ_idx >= num_problems) return;

  // Calculate G*J
  for (int i = threadIdx.x; i < data->NumContacts(); i += blockDim.x) {
    int J_row = i * 3;
    // do a simple matrix multiplication of 3x3 multiplied by 3 by
    // 3*num_velocities
    for (int a = 0; a < 3; a++) {
      for (int b = 0; b < 3 * data->NumVelocities(); b++) {
        data->G_J()(J_row + a, b) = 0;
        for (int c = 0; c < 3; c++) {
          data->G_J()(J_row + a, b) +=
              data->G(i)(a, c) * data->J()(J_row + c, b);
        }
      }
    }
  }

  // Calculate data->H() = J_transpose * (G*J)
  CalculateHessian(data);
}

__global__ void CalcNegGradKernel(SAPGPUData* data) {
  extern __shared__ double sums[];
  int equ_idx = blockIdx.x;
  int num_problems = data->NumProblems();
  int num_contacts = data->NumContacts();

  if (equ_idx >= num_problems) return;

  for (int i = threadIdx.x; i < 3 * data->NumVelocities(); i += blockDim.x) {
    double sum = 0.0;
    for (int j = 0; j < 3 * data->NumContacts(); j++) {
      sum += data->J()(j, i) * data->gamma_full()(j);
    }
    data->neg_grad()(i, 0) = -(data->momentum_gain()(i, 0) - sum);
  }
}

// ==========================================================================

void TestOneStepSapGPU(std::vector<SAPCPUData>& sap_cpu_data,
                       std::vector<double>& momentum_cost,
                       std::vector<double>& regularizer_cost,
                       std::vector<Eigen::MatrixXd>& hessian,
                       std::vector<Eigen::MatrixXd>& neg_grad,
                       int num_velocities, int num_contacts, int num_problems) {
  std::cout << "TestOneStepSapGPU with GPU called with " << num_problems
            << " problems" << std::endl;
  SAPGPUData sap_gpu_data;
  sap_gpu_data.MakeSAPGPUData(sap_cpu_data);

  // copy SAPGPUData to GPU
  SAPGPUData* d_sap_gpu_data;
  HANDLE_ERROR(hipMalloc(&d_sap_gpu_data, sizeof(SAPGPUData)));
  HANDLE_ERROR(hipMemcpy(d_sap_gpu_data, &sap_gpu_data, sizeof(SAPGPUData),
                          hipMemcpyHostToDevice));

  int threadsPerBlock = 32;

  // Evaluate Cost

  CalcMomentumCostKernel<<<num_problems, threadsPerBlock,
                           2048 * sizeof(double)>>>(d_sap_gpu_data);

  CalcRegularizerCostKernel<<<num_problems, threadsPerBlock,
                              2048 * sizeof(double)>>>(d_sap_gpu_data);

  HANDLE_ERROR(hipDeviceSynchronize());

  sap_gpu_data.RetriveMomentumCostToCPU(momentum_cost);
  sap_gpu_data.RetriveRegularizerCostToCPU(regularizer_cost);

  // Assemble Hessian
  // Calculate G*J
  CalcHessianKernel<<<num_problems, threadsPerBlock, 2048 * sizeof(double)>>>(
      d_sap_gpu_data);

  HANDLE_ERROR(hipDeviceSynchronize());

  // Assemble -grad
  CalcNegGradKernel<<<num_problems, threadsPerBlock, 2048 * sizeof(double)>>>(
      d_sap_gpu_data);

  HANDLE_ERROR(hipDeviceSynchronize());

  sap_gpu_data.RetriveHessianToCPU(hessian);
  sap_gpu_data.RetriveNegGradToCPU(neg_grad);
}

// ===========================================================================

// ===========================================================================
// Joe's Notes
// ===========================================================================
// Sets vc = J * v by modifying `data` __device__ void CalcConstraintVelocity(
//                   SAPGPUData * data) {
//   // vc = J*v
//   for (int i = threadIdx.x; i < data->NumContacts(); i += blockDim.x) {
//     data->v_vc(i) = data->v_J(i) * data->v_guess();
//   }
// }
// ===========================================================================