#include "hip/hip_runtime.h"
#include <stdio.h>

#include <iostream>

#include "cuda_cholesky.h"

// CUDA error handeling
// =====================
static void HandleError(hipError_t err, const char* file, int line) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))
// =====================
// Device function to perform Cholesky factorization
__device__ void CholeskyFactorizationFunc(double* M, double* L, int equ_idx,
                                          int thread_idx, size_t n,
                                          size_t num_stride) {
  Eigen::Map<Eigen::MatrixXd> d_M(M + equ_idx * n * n, n, n);
  Eigen::Map<Eigen::MatrixXd> d_L(L + equ_idx * n * n, n, n);

  for (int stride = 0; stride < num_stride; stride++) {
    int j_up = 31 + stride * 32;
    int j = thread_idx + stride * 32;
    // if (j >= n) return;

    for (int i = 0; i <= j_up; ++i) {
      __syncwarp();

      if (j < n && i <= j && i == j) {
        double sum = 0.0;
        for (int k = 0; k < i; ++k) {
          sum += d_L(i, k) * d_L(i, k);
        }
        d_L(i, i) = sqrt(d_M(i, i) - sum);
      }

      __syncwarp();

      if (j < n && i <= j && j > i) {
        double sum = 0.0;
        for (int k = 0; k < i; ++k) {
          sum += d_L(j, k) * d_L(i, k);
        }
        d_L(j, i) = (d_M(j, i) - sum) / d_L(i, i);
      }
    }
  }
}

__device__ void CholeskySolveForwardFunc(double* L, double* b, double* y,
                                         int equ_idx, int thread_idx, size_t n,
                                         size_t num_stride) {
  Eigen::Map<Eigen::MatrixXd> d_L(L + equ_idx * n * n, n, n);
  Eigen::Map<Eigen::VectorXd> d_b(b + equ_idx * n, n, 1);
  Eigen::Map<Eigen::VectorXd> d_y(y + equ_idx * n, n, 1);

  for (int stride = 0; stride < num_stride; stride++) {
    int j = thread_idx + stride * 32;
    int j_up = 31 + stride * 32;

    // Forward substitution to solve L * y = b
    double sum = 0.0;
    for (int i = 0; i <= j_up; ++i) {
      if (j < n && i <= j && i == j) {
        d_y(j) = (d_b(j) - sum) / d_L(j, j);
      }
      __syncwarp();

      if (j < n && i <= j && i < j) {
        sum += d_L(j, i) * d_y(i);
      }

      __syncwarp();
    }
  }
}

// Device function to perform backward substitution
__device__ void CholeskySolveBackwardFunc(double* L, double* y, double* x,
                                          int equ_idx, int thread_idx, size_t n,
                                          size_t num_stride) {
  Eigen::Map<Eigen::MatrixXd> d_L(L + equ_idx * n * n, n, n);
  Eigen::Map<Eigen::VectorXd> d_y(y + equ_idx * n, n, 1);
  Eigen::Map<Eigen::VectorXd> d_x(x + equ_idx * n, n, 1);

  for (int stride = 0; stride < num_stride; stride++) {
    int j = n - 1 - (thread_idx + stride * 32);
    int j_down = n - 1 - (31 + stride * 32);

    double sum = 0.0;
    for (int i = n - 1; i >= j_down; --i) {
      if (j >= 0 && i >= j && i == j) {
        d_x(j) = (d_y(j) - sum) / d_L(j, j);
      }
      __syncwarp();

      if (j >= 0 && i >= j && i > j) {
        sum += d_L(i, j) * d_x(i);
      }

      __syncwarp();
    }
  }
}

__global__ void CholeskySolveKernel(double* M, double* L, double* b, double* x,
                                    double* y, size_t num_problems, size_t n,
                                    size_t num_stride) {
  int equ_idx = blockIdx.x;
  int thread_idx = threadIdx.x;

  if (equ_idx >= num_problems) {
    return;
  }

  CholeskyFactorizationFunc(M, L, equ_idx, thread_idx, n, num_stride);
  __syncwarp();
  CholeskySolveForwardFunc(L, b, y, equ_idx, thread_idx, n, num_stride);
  __syncwarp();
  CholeskySolveBackwardFunc(L, y, x, equ_idx, thread_idx, n, num_stride);
  __syncwarp();
}

// Main solve function - including memory allocation, copy, and kernel calls
double MatrixSolve(std::vector<Eigen::MatrixXd>& M,
                   std::vector<Eigen::VectorXd>& b,
                   std::vector<Eigen::VectorXd>& x) {
  const int num_problems = M.size();
  const int n = b[0].size();

  double* x_result = new double[num_problems * n];

  // Allocate device arrays
  double *d_M, *d_b, *d_y, *d_x, *d_L;
  HANDLE_ERROR(hipMalloc((void**)&d_M, sizeof(double) * num_problems * n * n));
  HANDLE_ERROR(hipMalloc((void**)&d_L, sizeof(double) * num_problems * n * n));
  HANDLE_ERROR(hipMalloc((void**)&d_b, sizeof(double) * num_problems * n));
  HANDLE_ERROR(hipMalloc((void**)&d_x, sizeof(double) * num_problems * n));
  HANDLE_ERROR(hipMalloc((void**)&d_y, sizeof(double) * num_problems * n));

  // Set d_L and d_x to be 0
  HANDLE_ERROR(hipMemset(d_L, 0, sizeof(double) * num_problems * n * n));
  HANDLE_ERROR(hipMemset(d_x, 0, sizeof(double) * num_problems * n));
  HANDLE_ERROR(hipMemset(d_y, 0, sizeof(double) * num_problems * n));

  // Copy to device
  for (int i = 0; i < num_problems; ++i) {
    HANDLE_ERROR(hipMemcpy(d_M + i * n * n, M[i].data(),
                            sizeof(double) * n * n, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_b + i * n, b[i].data(), sizeof(double) * n,
                            hipMemcpyHostToDevice));
  }

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  int num_stride = (n + 31) / 32;

  // Matrix Cholesky factorization

  CholeskySolveKernel<<<num_problems, 32>>>(d_M, d_L, d_b, d_x, d_y,
                                            num_problems, n, num_stride);

  HANDLE_ERROR(hipGetLastError());
  HANDLE_ERROR(hipDeviceSynchronize());

  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float milliseconds = 0.0;
  hipEventElapsedTime(&milliseconds, start, stop);

  std::cout << "Elapsed time for Cholesky Solve: " << milliseconds << " ms\n";

  // Copy to host
  HANDLE_ERROR(hipMemcpy(x_result, d_x, sizeof(double) * num_problems * n,
                          hipMemcpyDeviceToHost));

  for (int i = 0; i < num_problems; ++i) {
    Eigen::Map<Eigen::VectorXd> x_result_i(x_result + i * n, n, 1);
    x[i] = x_result_i;
  }

  return 0;
}
