#include "hip/hip_runtime.h"
#include <stdio.h>

#include <iostream>

#include "cuda_cholesky.h"

static void HandleError(hipError_t err, const char* file, int line) {
  // CUDA error handeling from the "CUDA by example" book
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

__global__ void cholesky_factorization(double* M, double* L,
                                       size_t num_equations, size_t n,
                                       size_t offset) {
  int equ_idx = blockIdx.x;
  int thread_idx = threadIdx.x;

  if (equ_idx >= num_equations) {
    return;
  }

  Eigen::Map<Eigen::MatrixXd> d_M(M + equ_idx * n * n, n, n);
  Eigen::Map<Eigen::MatrixXd> d_L(L + equ_idx * n * n, n, n);

  int j = thread_idx + offset;
  if (j >= n) return;

  for (int i = 0; i <= j; ++i) {
    __syncwarp();

    if (i == j) {
      double sum = 0.0;
      for (int k = 0; k < i; ++k) {
        sum += d_L(i, k) * d_L(i, k);
      }
      d_L(i, i) = sqrt(d_M(i, i) - sum);
    }

    __syncwarp();

    if (j > i) {
      double sum = 0.0;
      for (int k = 0; k < i; ++k) {
        sum += d_L(j, k) * d_L(i, k);
      }
      d_L(j, i) = (d_M(j, i) - sum) / d_L(i, i);
    }
  }
}

__global__ void cholesky_solve_forward(double* L, double* b, double* x,
                                       double* y, size_t num_equations,
                                       size_t n, size_t offset) {
  int equ_idx = blockIdx.x;
  int thread_idx = threadIdx.x;

  if (equ_idx >= num_equations) {
    return;
  }

  Eigen::Map<Eigen::MatrixXd> d_L(L + equ_idx * n * n, n, n);
  Eigen::Map<Eigen::VectorXd> d_b(b + equ_idx * n, n, 1);
  Eigen::Map<Eigen::VectorXd> d_x(x + equ_idx * n, n, 1);
  Eigen::Map<Eigen::VectorXd> d_y(y + equ_idx * n, n, 1);

  int i = thread_idx + offset;
  if (i >= n) return;

  // Forward substitution to solve L * y = b

  double sum = 0.0;
  for (int j = 0; j <= i; ++j) {
    if (j == i) {
      d_y(i) = (d_b(i) - sum) / d_L(i, i);
    }
    __syncwarp();

    if (j < i) {
      sum += d_L(i, j) * d_y(j);
    }

    __syncwarp();
  }
}

__global__ void cholesky_solve_backward(double* L, double* b, double* x,
                                        double* y, size_t num_equations,
                                        size_t n, size_t offset) {
  int equ_idx = blockIdx.x;
  int thread_idx = threadIdx.x;

  if (equ_idx >= num_equations) {
    return;
  }

  Eigen::Map<Eigen::MatrixXd> d_L(L + equ_idx * n * n, n, n);
  Eigen::Map<Eigen::VectorXd> d_b(b + equ_idx * n, n, 1);
  Eigen::Map<Eigen::VectorXd> d_x(x + equ_idx * n, n, 1);
  Eigen::Map<Eigen::VectorXd> d_y(y + equ_idx * n, n, 1);

  int i = n - 1 - (thread_idx + offset);
  if (i < 0) return;

  double sum = 0.0;
  for (int j = n - 1; j >= i; --j) {
    if (j == i) {
      d_x(i) = (d_y(i) - sum) / d_L(i, i);
    }
    __syncwarp();

    if (j > i) {
      sum += d_L(j, i) * d_x(j);
    }

    __syncwarp();
  }
}

double matrix_solve(std::vector<Eigen::MatrixXd>& M,
                    std::vector<Eigen::VectorXd>& b,
                    std::vector<Eigen::VectorXd>& x) {
  const int num_equations = M.size();
  const int n = b[0].size();

  double* x_result = new double[num_equations * n];

  // Allocate device arrays
  double *d_M, *d_b, *d_y, *d_x, *d_L;
  HANDLE_ERROR(
      hipMalloc((void**)&d_M, sizeof(double) * num_equations * n * n));
  HANDLE_ERROR(
      hipMalloc((void**)&d_L, sizeof(double) * num_equations * n * n));
  HANDLE_ERROR(hipMalloc((void**)&d_b, sizeof(double) * num_equations * n));
  HANDLE_ERROR(hipMalloc((void**)&d_x, sizeof(double) * num_equations * n));
  HANDLE_ERROR(hipMalloc((void**)&d_y, sizeof(double) * num_equations * n));

  // Set d_L and d_x to be 0
  HANDLE_ERROR(hipMemset(d_L, 0, sizeof(double) * num_equations * n * n));
  HANDLE_ERROR(hipMemset(d_x, 1, sizeof(double) * num_equations * n));
  HANDLE_ERROR(hipMemset(d_y, 0, sizeof(double) * num_equations * n));

  // Copy to device
  for (int i = 0; i < num_equations; ++i) {
    HANDLE_ERROR(hipMemcpy(d_M + i * n * n, M[i].data(),
                            sizeof(double) * n * n, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_b + i * n, b[i].data(), sizeof(double) * n,
                            hipMemcpyHostToDevice));
  }

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  // Matrix Cholesky factorization
  int offset = 0;
  while (offset < n) {
    cholesky_factorization<<<num_equations, 32>>>(d_M, d_L, num_equations, n,
                                                  offset);
    offset += 32;

    HANDLE_ERROR(hipGetLastError());
    HANDLE_ERROR(hipDeviceSynchronize());
  }

  // Matrix Cholesky solve forward
  offset = 0;
  while (offset < n) {
    cholesky_solve_forward<<<num_equations, 32>>>(d_L, d_b, d_x, d_y,
                                                  num_equations, n, offset);
    offset += 32;

    HANDLE_ERROR(hipGetLastError());
    HANDLE_ERROR(hipDeviceSynchronize());
  }

  // Matrix Cholesky solve backward
  offset = 0;
  while (offset < n) {
    cholesky_solve_backward<<<num_equations, 32>>>(d_L, d_b, d_x, d_y,
                                                   num_equations, n, offset);
    offset += 32;

    HANDLE_ERROR(hipGetLastError());
    HANDLE_ERROR(hipDeviceSynchronize());
  }

  hipEventRecord(stop);

  hipEventSynchronize(stop);
  float milliseconds = 0.0;
  hipEventElapsedTime(&milliseconds, start, stop);

  std::cout << "Elapsed time for Cholesky Solve: " << milliseconds << " ms\n";

  // Copy to host
  HANDLE_ERROR(hipMemcpy(x_result, d_x, sizeof(double) * num_equations * n,
                          hipMemcpyDeviceToHost));

  for (int i = 0; i < num_equations; ++i) {
    Eigen::Map<Eigen::VectorXd> x_result_i(x_result + i * n, n, 1);
    std::cout << "||M*x - b||: " << (M[i] * x_result_i - b[i]).norm()
              << std::endl;
  }

  return 0;
}
