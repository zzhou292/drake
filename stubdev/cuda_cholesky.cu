#include "hip/hip_runtime.h"
#include <stdio.h>

#include <iostream>

#include "cuda_cholesky.cuh"
#include "cuda_cholesky.h"
// CUDA error handeling
// =====================
static void HandleError(hipError_t err, const char* file, int line) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

__global__ void CholeskySolveKernel(float* M, float* L, float* b, float* x,
                                    float* y, size_t num_problems, size_t n) {
  int equ_idx = blockIdx.x;
  int thread_idx = threadIdx.x;

  if (equ_idx >= num_problems) {
    return;
  }

  Eigen::Map<Eigen::MatrixXf> d_M(M + equ_idx * n * n, n, n);
  Eigen::Map<Eigen::MatrixXf> d_L(L + equ_idx * n * n, n, n);
  Eigen::Map<Eigen::MatrixXf> d_b(b + equ_idx * n, n, 1);
  Eigen::Map<Eigen::MatrixXf> d_x(x + equ_idx * n, n, 1);
  Eigen::Map<Eigen::MatrixXf> d_y(y + equ_idx * n, n, 1);

  int num_stride = (n + 31) / 32;

  CholeskyFactorizationFunc(d_M, d_L, equ_idx, thread_idx, n, num_stride);
  __syncwarp();
  CholeskySolveForwardFunc(d_L, d_b, d_y, equ_idx, thread_idx, n, num_stride);
  __syncwarp();
  CholeskySolveBackwardFunc(d_L, d_y, d_x, equ_idx, thread_idx, n, num_stride);
  __syncwarp();
}

// Main solve function - including memory allocation, copy, and kernel calls
float MatrixSolve(std::vector<Eigen::MatrixXf>& M,
                  std::vector<Eigen::MatrixXf>& b,
                  std::vector<Eigen::MatrixXf>& x) {
  const int num_problems = M.size();
  const int n = b[0].size();

  float* x_result = new float[num_problems * n];

  // Allocate device arrays
  float *d_M, *d_b, *d_y, *d_x, *d_L;
  HANDLE_ERROR(hipMalloc((void**)&d_M, sizeof(float) * num_problems * n * n));
  HANDLE_ERROR(hipMalloc((void**)&d_L, sizeof(float) * num_problems * n * n));
  HANDLE_ERROR(hipMalloc((void**)&d_b, sizeof(float) * num_problems * n));
  HANDLE_ERROR(hipMalloc((void**)&d_x, sizeof(float) * num_problems * n));
  HANDLE_ERROR(hipMalloc((void**)&d_y, sizeof(float) * num_problems * n));

  // Set d_L and d_x to be 0
  HANDLE_ERROR(hipMemset(d_L, 0, sizeof(float) * num_problems * n * n));
  HANDLE_ERROR(hipMemset(d_x, 0, sizeof(float) * num_problems * n));
  HANDLE_ERROR(hipMemset(d_y, 0, sizeof(float) * num_problems * n));

  // Copy to device
  for (int i = 0; i < num_problems; ++i) {
    HANDLE_ERROR(hipMemcpy(d_M + i * n * n, M[i].data(), sizeof(float) * n * n,
                            hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_b + i * n, b[i].data(), sizeof(float) * n,
                            hipMemcpyHostToDevice));
  }

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  // Matrix Cholesky factorization
  CholeskySolveKernel<<<num_problems, 32>>>(d_M, d_L, d_b, d_x, d_y,
                                            num_problems, n);

  HANDLE_ERROR(hipGetLastError());
  HANDLE_ERROR(hipDeviceSynchronize());

  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float milliseconds = 0.0;
  hipEventElapsedTime(&milliseconds, start, stop);

  std::cout << "Elapsed time for Cholesky Solve: " << milliseconds << " ms\n";

  // Copy to host
  HANDLE_ERROR(hipMemcpy(x_result, d_x, sizeof(float) * num_problems * n,
                          hipMemcpyDeviceToHost));

  for (int i = 0; i < num_problems; ++i) {
    Eigen::Map<Eigen::MatrixXf> x_result_i(x_result + i * n, n, 1);
    x[i] = x_result_i;
  }

  return 0;
}
