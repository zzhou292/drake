#include "hip/hip_runtime.h"
#include <stdio.h>

#include <iostream>

#include "cuda_gpu_collision.cuh"
#include "cuda_gpu_collision.h"

#ifndef dt
#define dt 0.01
#endif

#ifndef gravity
#define gravity -9.81
#endif

// Device function to check Sphere-Sphere collision
__device__ CollisionData CheckSphereCollision(const Sphere& a,
                                              const Sphere& b) {
  CollisionData data = {
      false, {0, 0, 0}, {0, 0, 0}, 0, Eigen::Matrix3d::Zero()};

  Eigen::Vector3d dist = a.center - b.center;
  double distSquared =
      dist(0) * dist(0) + dist(1) * dist(1) + dist(2) * dist(2);
  double distLength = sqrt(distSquared);
  double radiusSum = a.radius + b.radius;

  dist.normalize();

  if (distSquared <= (radiusSum * radiusSum)) {
    data.isColliding = true;
    // Calculate collision normal
    data.nhat_BA_W = dist;
    // Normalize the collision normal

    data.nhat_BA_W.normalize();
    // Calculate collision points
    Eigen::Vector3d midpoint;
    midpoint(0) = (a.center(0) + b.center(0)) / 2;
    midpoint(1) = (a.center(1) + b.center(1)) / 2;
    midpoint(2) = (a.center(2) + b.center(2)) / 2;

    data.phi0 = -(distLength - radiusSum);  // sign convention

    data.p_WC = midpoint;

    // Get collision frame matrix
    // Random vector v is default to {1.0, 1.0, 1.0}
    Eigen::Vector3d v(1.0, 1.0, 1.0);
    v.normalize();

    double y_hat_temp = v.dot(data.nhat_BA_W);
    Eigen::Vector3d y_hat = v - y_hat_temp * data.nhat_BA_W;
    y_hat.normalize();
    Eigen::Vector3d x_hat = y_hat.cross(data.nhat_BA_W);

    data.R(0, 0) = x_hat(0);           // x of x-axis
    data.R(0, 1) = x_hat(1);           // y of x-axis
    data.R(0, 2) = x_hat(2);           // z of x-axis
    data.R(1, 0) = y_hat(0);           // x of y-axis
    data.R(1, 1) = y_hat(1);           // y of y-axis
    data.R(1, 2) = y_hat(2);           // z of y-axis
    data.R(2, 0) = data.nhat_BA_W(0);  // x of z-axis
    data.R(2, 1) = data.nhat_BA_W(1);  // y of z-axis
    data.R(2, 2) = data.nhat_BA_W(2);  // z of z-axis

    data.vn = -(a.velocity - b.velocity)
                   .dot(data.nhat_BA_W);  // negative for departing, positive
                                          // for approaching
  } else {
    data.isColliding = false;
  }

  return data;
}

// Kernel to detect collisions between Spheres
__global__ void DetectSphereCollisions(const Sphere* spheres, int numProblems,
                                       int numSpheres,
                                       CollisionData* collisionMatrix) {
  int idx = threadIdx.x;
  int p_idx = blockIdx.x;

  for (int j = idx; j < numSpheres; j += blockDim.x) {
    if (j < numSpheres) {
      for (int k = j + 1; k < numSpheres; k++) {
        collisionMatrix[(p_idx * numSpheres * numSpheres) + j * numSpheres +
                        k] =
            CheckSphereCollision(spheres[p_idx * numSpheres + j],
                                 spheres[p_idx * numSpheres + k]);
      }
    }
  }
  __syncwarp();
}

// Kernel to detect collisions between Spheres
__global__ void ConstructJacobianGamma(
    const Sphere* spheres, int numProblems, int numSpheres,
    CollisionData* collisionMatrix, double* d_jacobian, int* d_num_collisions,
    double* d_phi0, double* d_contact_stiffness, double* d_contact_damping) {
  int idx = threadIdx.x;
  int p_idx = blockIdx.x;

  Eigen::Map<Eigen::MatrixXd> full_jacobian(
      d_jacobian +
          blockIdx.x * (numSpheres * 3) * (numSpheres * numSpheres * 3),
      numSpheres * numSpheres * 3, numSpheres * 3);
  Eigen::Map<Eigen::VectorXd> contact_stiffness(
      d_contact_stiffness + blockIdx.x * numSpheres * numSpheres,
      numSpheres * numSpheres, 1);
  Eigen::Map<Eigen::VectorXd> contact_damping(
      d_contact_damping + blockIdx.x * numSpheres * numSpheres,
      numSpheres * numSpheres, 1);

  for (int j = idx; j < numSpheres; j += blockDim.x) {
    if (j < numSpheres) {
      for (int k = j + 1; k < numSpheres; k++) {
        if (collisionMatrix[(p_idx * numSpheres * numSpheres) + j * numSpheres +
                            k]
                .isColliding) {
          int collision_idx = atomicAdd(&d_num_collisions[p_idx], 1);

          // update the harmonic mean of contact stiffness
          contact_stiffness[collision_idx] =
              (2 * spheres[p_idx * numSpheres + j].stiffness *
               spheres[p_idx * numSpheres + k].stiffness) /
              (spheres[p_idx * numSpheres + j].stiffness +
               spheres[p_idx * numSpheres + k].stiffness);

          // update the harmonic mean of contact damping
          contact_damping[collision_idx] =
              (2 * spheres[p_idx * numSpheres + j].damping *
               spheres[p_idx * numSpheres + k].damping) /
              (spheres[p_idx * numSpheres + j].damping +
               spheres[p_idx * numSpheres + k].damping);

          // construct Jacobian matrix
          full_jacobian.block<3, 3>(collision_idx * 3, j * 3) =
              collisionMatrix[(p_idx * numSpheres * numSpheres) +
                              j * numSpheres + k]
                  .R *
              Eigen::MatrixXd::Identity(3, 3);
          full_jacobian.block<3, 3>(collision_idx * 3, k * 3) =
              collisionMatrix[(p_idx * numSpheres * numSpheres) +
                              j * numSpheres + k]
                  .R *
              -Eigen::MatrixXd::Identity(3, 3);

          // add data to phi0
          d_phi0[p_idx * numSpheres * numSpheres + collision_idx] =
              collisionMatrix[(p_idx * numSpheres * numSpheres) +
                              j * numSpheres + k]
                  .phi0;
        }
      }
    }
  }
  __syncwarp();
}

__global__ void ConstructDynamicMatrixVelocityVector(
    const Sphere* spheres, int numProblems, int numSpheres,
    double* d_dynamic_matrix, double* d_velocity_vector) {
  int idx = threadIdx.x;
  int p_idx = blockIdx.x;

  Eigen::Map<Eigen::MatrixXd> dynamic_matrix(
      d_dynamic_matrix + blockIdx.x * numSpheres * 3 * numSpheres * 3,
      numSpheres * 3, numSpheres * 3);
  Eigen::Map<Eigen::VectorXd> velocity_vector(
      d_velocity_vector + blockIdx.x * numSpheres * 3, numSpheres * 3, 1);

  for (int j = idx; j < numSpheres; j += blockDim.x) {
    if (j < numSpheres) {
      dynamic_matrix.block<3, 3>(j * 3, j * 3) =
          spheres[p_idx * numSpheres + j].mass *
          Eigen::MatrixXd::Identity(3, 3);
      velocity_vector.block<3, 1>(j * 3, 0) =
          spheres[p_idx * numSpheres + j].velocity;
    }
  }

  __syncwarp();
}

__global__ void CalculateFreeMotionVelocity(const Sphere* spheres,
                                            int numProblems, int numSpheres,
                                            double* d_velocity_vector,
                                            double* d_v_star) {
  int idx = threadIdx.x;
  int p_idx = blockIdx.x;

  Eigen::Map<Eigen::VectorXd> velocity_vector(
      d_velocity_vector + blockIdx.x * numSpheres * 3, numSpheres * 3, 1);
  Eigen::Map<Eigen::VectorXd> v_star(d_v_star + blockIdx.x * numSpheres * 3,
                                     numSpheres * 3, 1);

  for (int j = idx; j < numSpheres; j += blockDim.x) {
    if (j < numSpheres) {
      v_star.block<3, 1>(j * 3, 0) =
          velocity_vector.block<3, 1>(j * 3, 0) + dt * Eigen::Vector3d(0, 0, 0);
    }
  }

  __syncwarp();
}

void CollisionGPUData::CollisionEngine(const int numProblems,
                                       const int numSpheres) {
  // Kernel launches
  int threadsPerBlock = 32;
  int blocksPerGridSpheres = numProblems;
  DetectSphereCollisions<<<blocksPerGridSpheres, threadsPerBlock>>>(
      this->GetSpherePtr(), numProblems, numSpheres,
      this->GetCollisionMatrixPtr());
  HANDLE_ERROR(hipDeviceSynchronize());

  // Construct Jacobian matrix and Gamma vector
  ConstructJacobianGamma<<<blocksPerGridSpheres, threadsPerBlock>>>(
      this->GetSpherePtr(), numProblems, numSpheres,
      this->GetCollisionMatrixPtr(), this->GetJacobianPtr(),
      this->GetNumCollisionsPtr(), this->GetPhi0Ptr(),
      this->GetContactStiffnessPtr(), this->GetContactDampingPtr());
  HANDLE_ERROR(hipDeviceSynchronize());

  // Construct Dynamic matrix
  ConstructDynamicMatrixVelocityVector<<<blocksPerGridSpheres,
                                         threadsPerBlock>>>(
      this->GetSpherePtr(), numProblems, numSpheres,
      this->GetDynamicMatrixPtr(), this->GetVelocityVectorPtr());
  HANDLE_ERROR(hipDeviceSynchronize());

  // Calculate free motion velocity vector Dynamic matrix
  CalculateFreeMotionVelocity<<<blocksPerGridSpheres, threadsPerBlock>>>(
      this->GetSpherePtr(), numProblems, numSpheres,
      this->GetVelocityVectorPtr(), this->GetVStarPtr());
  HANDLE_ERROR(hipDeviceSynchronize());
}

__global__ void IntegrateExplicitEulerKernel(Sphere* spheres, int numProblems,
                                             int numSpheres,
                                             double* d_velocity_vector) {
  int idx = threadIdx.x;
  int p_idx = blockIdx.x;

  Eigen::Map<Eigen::VectorXd> velocity_vector(
      d_velocity_vector + blockIdx.x * numSpheres * 3, numSpheres * 3, 1);

  for (int j = idx; j < numSpheres; j += blockDim.x) {
    spheres[p_idx * numSpheres + j].velocity =
        velocity_vector.block<3, 1>(j * 3, 0);
    spheres[p_idx * numSpheres + j].center =
        spheres[p_idx * numSpheres + j].center +
        dt * velocity_vector.block<3, 1>(j * 3, 0);
  }

  __syncwarp();
}

// an eplicit euler to update the position based on velocity_vector
void CollisionGPUData::IntegrateExplicitEuler(const int numProblems,
                                              const int numSpheres) {
  // Kernel launches
  int threadsPerBlock = 32;
  int blocksPerGridSpheres = numProblems;
  IntegrateExplicitEulerKernel<<<blocksPerGridSpheres, threadsPerBlock>>>(
      d_spheres, numProblems, numSpheres, this->GetVelocityVectorPtr());
  HANDLE_ERROR(hipDeviceSynchronize());
}