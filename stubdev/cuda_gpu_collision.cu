#include "hip/hip_runtime.h"
#include <stdio.h>

#include <iostream>

#include "cuda_gpu_collision.h"

#ifndef dt
#define dt 0.002
#endif

#ifndef gravity
#define gravity -9.81
#endif

// CUDA error handeling
// =====================
static void HandleError(hipError_t err, const char* file, int line) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))
// =====================

// Device function to check Sphere-Sphere collision
__device__ CollisionData CheckSphereCollision(const Sphere& a,
                                              const Sphere& b) {
  CollisionData data = {
      false, {0, 0, 0}, {0, 0, 0}, 0, Eigen::Matrix3d::Zero()};

  Eigen::Vector3d dist = a.center - b.center;
  double distSquared =
      dist(0) * dist(0) + dist(1) * dist(1) + dist(2) * dist(2);
  double distLength = sqrt(distSquared);
  double radiusSum = a.radius + b.radius;

  dist.normalize();

  if (distSquared <= (radiusSum * radiusSum)) {
    data.isColliding = true;
    // Calculate collision normal
    data.nhat_BA_W = dist;
    // Normalize the collision normal

    data.nhat_BA_W.normalize();
    // Calculate collision points
    Eigen::Vector3d midpoint;
    midpoint(0) = (a.center(0) + b.center(0)) / 2;
    midpoint(1) = (a.center(1) + b.center(1)) / 2;
    midpoint(2) = (a.center(2) + b.center(2)) / 2;

    data.phi0 = -(distLength - radiusSum);  // sign convention

    data.p_WC = midpoint;

    // Get collision frame matrix
    // Random vector v is default to {1.0, 1.0, 1.0}
    Eigen::Vector3d v(1.0, 1.0, 1.0);
    v.normalize();

    double y_hat_temp = v.dot(data.nhat_BA_W);
    Eigen::Vector3d y_hat = v - y_hat_temp * data.nhat_BA_W;
    y_hat.normalize();
    Eigen::Vector3d x_hat = y_hat.cross(data.nhat_BA_W);

    data.R(0, 0) = x_hat(0);           // x of x-axis
    data.R(0, 1) = x_hat(1);           // y of x-axis
    data.R(0, 2) = x_hat(2);           // z of x-axis
    data.R(1, 0) = y_hat(0);           // x of y-axis
    data.R(1, 1) = y_hat(1);           // y of y-axis
    data.R(1, 2) = y_hat(2);           // z of y-axis
    data.R(2, 0) = data.nhat_BA_W(0);  // x of z-axis
    data.R(2, 1) = data.nhat_BA_W(1);  // y of z-axis
    data.R(2, 2) = data.nhat_BA_W(2);  // z of z-axis

    data.vn = -(a.velocity - b.velocity)
                   .dot(data.nhat_BA_W);  // negative for departing, positive
                                          // for approaching
  } else {
    data.isColliding = false;
  }

  return data;
}

// Kernel to detect collisions between Spheres
__global__ void DetectSphereCollisions(const Sphere* spheres, int numProblems,
                                       int numSpheres,
                                       CollisionData* collisionMatrix) {
  int idx = threadIdx.x;
  int p_idx = blockIdx.x;

  for (int j = idx; j < numSpheres; j += blockDim.x) {
    if (j < numSpheres) {
      for (int k = j + 1; k < numSpheres; k++) {
        collisionMatrix[(p_idx * numSpheres * numSpheres) + j * numSpheres +
                        k] =
            CheckSphereCollision(spheres[p_idx * numSpheres + j],
                                 spheres[p_idx * numSpheres + k]);
      }
    }
  }
  __syncwarp();
}

// Kernel to detect collisions between Spheres
__global__ void ConstructJacobianGamma(
    const Sphere* spheres, int numProblems, int numSpheres,
    CollisionData* collisionMatrix, double* d_jacobian, int* d_num_collisions,
    double* d_phi0, double* d_contact_stiffness, double* d_contact_damping) {
  int idx = threadIdx.x;
  int p_idx = blockIdx.x;

  Eigen::Map<Eigen::MatrixXd> full_jacobian(
      d_jacobian +
          blockIdx.x * (numSpheres * 3) * (numSpheres * numSpheres * 3),
      numSpheres * numSpheres * 3, numSpheres * 3);
  Eigen::Map<Eigen::VectorXd> contact_stiffness(
      d_contact_stiffness + blockIdx.x * numSpheres * numSpheres,
      numSpheres * numSpheres, 1);
  Eigen::Map<Eigen::VectorXd> contact_damping(
      d_contact_damping + blockIdx.x * numSpheres * numSpheres,
      numSpheres * numSpheres, 1);

  for (int j = idx; j < numSpheres; j += blockDim.x) {
    if (j < numSpheres) {
      for (int k = j + 1; k < numSpheres; k++) {
        if (collisionMatrix[(p_idx * numSpheres * numSpheres) + j * numSpheres +
                            k]
                .isColliding) {
          int collision_idx = atomicAdd(&d_num_collisions[p_idx], 1);

          // update the harmonic mean of contact stiffness
          contact_stiffness[collision_idx] =
              (2 * spheres[p_idx * numSpheres + j].stiffness *
               spheres[p_idx * numSpheres + k].stiffness) /
              (spheres[p_idx * numSpheres + j].stiffness +
               spheres[p_idx * numSpheres + k].stiffness);

          // update the harmonic mean of contact damping
          contact_damping[collision_idx] =
              (2 * spheres[p_idx * numSpheres + j].damping *
               spheres[p_idx * numSpheres + k].damping) /
              (spheres[p_idx * numSpheres + j].damping +
               spheres[p_idx * numSpheres + k].damping);

          // construct Jacobian matrix
          full_jacobian.block<3, 3>(collision_idx * 3, j * 3) =
              collisionMatrix[(p_idx * numSpheres * numSpheres) +
                              j * numSpheres + k]
                  .R *
              Eigen::MatrixXd::Identity(3, 3);
          full_jacobian.block<3, 3>(collision_idx * 3, k * 3) =
              collisionMatrix[(p_idx * numSpheres * numSpheres) +
                              j * numSpheres + k]
                  .R *
              -Eigen::MatrixXd::Identity(3, 3);

          // add data to phi0
          d_phi0[p_idx * numSpheres * numSpheres + collision_idx] =
              collisionMatrix[(p_idx * numSpheres * numSpheres) +
                              j * numSpheres + k]
                  .phi0;
        }
      }
    }
  }
  __syncwarp();
}

__global__ void ConstructDynamicMatrixVelocityVector(
    const Sphere* spheres, int numProblems, int numSpheres,
    double* d_dynamic_matrix, double* d_velocity_vector) {
  int idx = threadIdx.x;
  int p_idx = blockIdx.x;

  Eigen::Map<Eigen::MatrixXd> dynamic_matrix(
      d_dynamic_matrix + blockIdx.x * numSpheres * 3 * numSpheres * 3,
      numSpheres * 3, numSpheres * 3);
  Eigen::Map<Eigen::VectorXd> velocity_vector(
      d_velocity_vector + blockIdx.x * numSpheres * 3, numSpheres * 3, 1);

  for (int j = idx; j < numSpheres; j += blockDim.x) {
    if (j < numSpheres) {
      dynamic_matrix.block<3, 3>(j * 3, j * 3) =
          spheres[p_idx * numSpheres + j].mass *
          Eigen::MatrixXd::Identity(3, 3);
      velocity_vector.block<3, 1>(j * 3, 0) =
          spheres[p_idx * numSpheres + j].velocity;
    }
  }

  __syncwarp();
}

__global__ void CalculateFreeMotionVelocity(const Sphere* spheres,
                                            int numProblems, int numSpheres,
                                            double* d_velocity_vector,
                                            double* d_v_star) {
  int idx = threadIdx.x;
  int p_idx = blockIdx.x;

  Eigen::Map<Eigen::VectorXd> velocity_vector(
      d_velocity_vector + blockIdx.x * numSpheres * 3, numSpheres * 3, 1);
  Eigen::Map<Eigen::VectorXd> v_star(d_v_star + blockIdx.x * numSpheres * 3,
                                     numSpheres * 3, 1);

  for (int j = idx; j < numSpheres; j += blockDim.x) {
    if (j < numSpheres) {
      v_star.block<3, 1>(j * 3, 0) =
          velocity_vector.block<3, 1>(j * 3, 0) + dt * Eigen::Vector3d(0, 0, 0);
    }
  }

  __syncwarp();
}

void CollisionEngine(Sphere* h_spheres, const int numProblems,
                     const int numSpheres,
                     CollisionData* h_collisionMatrixSpheres,
                     double* h_jacobian, int* h_num_collisions,
                     double* h_dynamic_matrix, double* h_velocity_vector,
                     double* h_v_star, double* h_phi0,
                     double* h_contact_stiffness, double* h_contact_damping) {
  // Device memory allocations
  Sphere* d_spheres;
  CollisionData* d_collisionMatrixSpheres;

  int* d_num_collisions;
  double* d_jacobian;
  double* d_dynamic_matrix;  // for now, we deal with 3DOF per body, so A matrix
                             // is 3*numsphere x 3*numsphere
  double* d_velocity_vector;  // for now, we deal with 3DOF per body, so
                              // velocity vector is 3*numsphere x 1
  double* d_v_star;
  double* d_phi0;
  double* d_contact_stiffness;
  double* d_contact_damping;

  std::cout << "haha 0" << std::endl;
  std::cout << "sizeof(Spheres): " << sizeof(Sphere) << std::endl;
  std::cout << "sizeof(double)  " << sizeof(double) << std::endl;
  std::cout << "numProblems: " << numProblems << std::endl;

  double* d_test;
  HANDLE_ERROR(hipMalloc((void**)&d_test, sizeof(double)));

  std::cout << "end test call" << std::endl;

  HANDLE_ERROR(hipMalloc((void**)&d_spheres,
                          numProblems * numSpheres * sizeof(Sphere)));
  std::cout << "haha 1" << std::endl;
  HANDLE_ERROR(hipMalloc(
      (void**)&d_collisionMatrixSpheres,
      numProblems * numSpheres * numSpheres * sizeof(CollisionData)));
  std::cout << "haha 2" << std::endl;
  HANDLE_ERROR(hipMalloc((void**)&d_jacobian,
                          numProblems * sizeof(double) * (numSpheres * 3) *
                              numSpheres * numSpheres * 3));
  std::cout << "haha 3" << std::endl;
  HANDLE_ERROR(
      hipMalloc((void**)&d_num_collisions, numProblems * sizeof(int)));
  HANDLE_ERROR(hipMalloc(
      (void**)&d_dynamic_matrix,
      numProblems * sizeof(double) * numSpheres * 3 * numSpheres * 3));
  HANDLE_ERROR(hipMalloc((void**)&d_velocity_vector,
                          numProblems * sizeof(double) * numSpheres * 3));
  HANDLE_ERROR(hipMalloc((void**)&d_v_star,
                          numProblems * sizeof(double) * numSpheres * 3));
  HANDLE_ERROR(hipMalloc(
      (void**)&d_phi0, numProblems * sizeof(double) * numSpheres * numSpheres));
  HANDLE_ERROR(
      hipMalloc((void**)&d_contact_stiffness,
                 numProblems * sizeof(double) * numSpheres * numSpheres));
  HANDLE_ERROR(
      hipMalloc((void**)&d_contact_damping,
                 numProblems * sizeof(double) * numSpheres * numSpheres));

  // Copy data to device
  HANDLE_ERROR(hipMemcpy(d_spheres, h_spheres,
                          numProblems * numSpheres * sizeof(Sphere),
                          hipMemcpyHostToDevice));

  // Set jacobian, num_collisions, full gamma vector to zero
  HANDLE_ERROR(hipMemset(d_jacobian, 0,
                          numProblems * sizeof(double) * (numSpheres * 3) *
                              numSpheres * numSpheres * 3));
  HANDLE_ERROR(hipMemset(d_num_collisions, 0, numProblems * sizeof(int)));
  HANDLE_ERROR(hipMemset(
      d_dynamic_matrix, 0,
      numProblems * sizeof(double) * numSpheres * 3 * numSpheres * 3));
  HANDLE_ERROR(hipMemset(d_velocity_vector, 0,
                          numProblems * sizeof(double) * numSpheres * 3));
  HANDLE_ERROR(
      hipMemset(d_v_star, 0, numProblems * sizeof(double) * numSpheres * 3));
  HANDLE_ERROR(hipMemset(
      d_phi0, 0, numProblems * sizeof(double) * numSpheres * numSpheres));
  HANDLE_ERROR(
      hipMemset(d_contact_stiffness, 0,
                 numProblems * sizeof(double) * numSpheres * numSpheres));
  HANDLE_ERROR(
      hipMemset(d_contact_damping, 0,
                 numProblems * sizeof(double) * numSpheres * numSpheres));

  // Kernel launches
  int threadsPerBlock = 32;
  int blocksPerGridSpheres = numProblems;
  DetectSphereCollisions<<<blocksPerGridSpheres, threadsPerBlock>>>(
      d_spheres, numProblems, numSpheres, d_collisionMatrixSpheres);
  HANDLE_ERROR(hipDeviceSynchronize());

  // Construct Jacobian matrix and Gamma vector
  ConstructJacobianGamma<<<blocksPerGridSpheres, threadsPerBlock>>>(
      d_spheres, numProblems, numSpheres, d_collisionMatrixSpheres, d_jacobian,
      d_num_collisions, d_phi0, d_contact_stiffness, d_contact_damping);
  HANDLE_ERROR(hipDeviceSynchronize());

  // Construct Dynamic matrix
  ConstructDynamicMatrixVelocityVector<<<blocksPerGridSpheres,
                                         threadsPerBlock>>>(
      d_spheres, numProblems, numSpheres, d_dynamic_matrix, d_velocity_vector);
  HANDLE_ERROR(hipDeviceSynchronize());

  // Calculate free motion velocity vector Dynamic matrix
  CalculateFreeMotionVelocity<<<blocksPerGridSpheres, threadsPerBlock>>>(
      d_spheres, numProblems, numSpheres, d_velocity_vector, d_v_star);
  HANDLE_ERROR(hipDeviceSynchronize());

  // Copy results back to host
  HANDLE_ERROR(
      hipMemcpy(h_collisionMatrixSpheres, d_collisionMatrixSpheres,
                 numProblems * numSpheres * numSpheres * sizeof(CollisionData),
                 hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(h_jacobian, d_jacobian,
                          numProblems * sizeof(double) * (numSpheres * 3) *
                              numSpheres * numSpheres * 3,
                          hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(h_num_collisions, d_num_collisions,
                          numProblems * sizeof(int), hipMemcpyDeviceToHost));
  HANDLE_ERROR(
      hipMemcpy(h_dynamic_matrix, d_dynamic_matrix,
                 numProblems * sizeof(double) * numSpheres * 3 * numSpheres * 3,
                 hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(h_velocity_vector, d_velocity_vector,
                          numProblems * sizeof(double) * numSpheres * 3,
                          hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(h_v_star, d_v_star,
                          numProblems * sizeof(double) * numSpheres * 3,
                          hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(
      h_phi0, d_phi0, numProblems * sizeof(double) * numSpheres * numSpheres,
      hipMemcpyDeviceToHost));
  HANDLE_ERROR(
      hipMemcpy(h_contact_stiffness, d_contact_stiffness,
                 numProblems * sizeof(double) * numSpheres * numSpheres,
                 hipMemcpyDeviceToHost));
  HANDLE_ERROR(
      hipMemcpy(h_contact_damping, d_contact_damping,
                 numProblems * sizeof(double) * numSpheres * numSpheres,
                 hipMemcpyDeviceToHost));

  // Free device memory

  hipFree(d_dynamic_matrix);
  hipFree(d_velocity_vector);
  hipFree(d_v_star);
  hipFree(d_phi0);
  hipFree(d_contact_damping);
  hipFree(d_contact_stiffness);
  hipFree(d_spheres);
  hipFree(d_collisionMatrixSpheres);
  hipFree(d_jacobian);
  hipFree(d_num_collisions);
}
