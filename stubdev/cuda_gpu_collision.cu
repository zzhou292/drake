#include "hip/hip_runtime.h"
#include <stdio.h>

#include <iostream>

#include "cuda_gpu_collision.h"

// CUDA error handeling
// =====================
static void HandleError(hipError_t err, const char* file, int line) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))
// =====================

// Device function to check Sphere-Sphere collision
__device__ CollisionData CheckSphereCollision(const Sphere& a,
                                              const Sphere& b) {
  CollisionData data = {
      false, {0, 0, 0}, {0, 0, 0}, 0, Eigen::Matrix3d::Zero()};

  Eigen::Vector3d dist = a.center - b.center;
  double distSquared =
      dist(0) * dist(0) + dist(1) * dist(1) + dist(2) * dist(2);
  double distLength = sqrt(distSquared);
  double radiusSum = a.radius + b.radius;

  dist.normalize();

  if (distSquared <= (radiusSum * radiusSum)) {
    data.isColliding = true;
    // Calculate collision normal
    data.nhat_BA_W = dist;
    // Normalize the collision normal

    data.nhat_BA_W.normalize();
    // Calculate collision points
    Eigen::Vector3d midpoint;
    midpoint(0) = (a.center(0) + b.center(0)) / 2;
    midpoint(1) = (a.center(1) + b.center(1)) / 2;
    midpoint(2) = (a.center(2) + b.center(2)) / 2;

    data.phi0 = distLength - radiusSum;

    data.p_WC = midpoint;

    // Get collision frame matrix
    // Random vector v is default to {1.0, 1.0, 1.0}
    Eigen::Vector3d v(1.0, 1.0, 1.0);
    v.normalize();

    double y_hat_temp = v.dot(data.nhat_BA_W);
    Eigen::Vector3d y_hat = v - y_hat_temp * data.nhat_BA_W;
    y_hat.normalize();
    Eigen::Vector3d x_hat = y_hat.cross(data.nhat_BA_W);

    data.R(0, 0) = x_hat(0);           // x of x-axis
    data.R(0, 1) = x_hat(1);           // y of x-axis
    data.R(0, 2) = x_hat(2);           // z of x-axis
    data.R(1, 0) = y_hat(0);           // x of y-axis
    data.R(1, 1) = y_hat(1);           // y of y-axis
    data.R(1, 2) = y_hat(2);           // z of y-axis
    data.R(2, 0) = data.nhat_BA_W(0);  // x of z-axis
    data.R(2, 1) = data.nhat_BA_W(1);  // y of z-axis
    data.R(2, 2) = data.nhat_BA_W(2);  // z of z-axis

    data.vn = (a.velocity - b.velocity).dot(data.nhat_BA_W);
  } else {
    data.isColliding = false;
  }

  return data;
}

// Kernel to detect collisions between Spheres
__global__ void DetectSphereCollisions(const Sphere* spheres, int numProblems,
                                       int numSpheres,
                                       CollisionData* collisionMatrix) {
  int idx = threadIdx.x;
  int p_idx = blockIdx.x;

  int num_stride = numSpheres / 32 + 1;

  for (int j = idx; j < numSpheres; j += blockDim.x) {
    if (j < numSpheres) {
      for (int k = j + 1; k < numSpheres; k++) {
        collisionMatrix[(p_idx * numSpheres * numSpheres) + j * numSpheres +
                        k] =
            CheckSphereCollision(spheres[p_idx * numSpheres + j],
                                 spheres[p_idx * numSpheres + k]);
      }
    }
  }
  __syncwarp();
}

void CollisionEngine(Sphere* h_spheres, const int numProblems,
                     const int numSpheres,
                     CollisionData* h_collisionMatrixSpheres) {
  // Device memory allocations
  Sphere* d_spheres;
  CollisionData* d_collisionMatrixSpheres;

  HANDLE_ERROR(hipMalloc((void**)&d_spheres,
                          numProblems * numSpheres * sizeof(Sphere)));
  HANDLE_ERROR(hipMalloc(
      (void**)&d_collisionMatrixSpheres,
      numProblems * numSpheres * numSpheres * sizeof(CollisionData)));
  // Copy data to device
  HANDLE_ERROR(hipMemcpy(d_spheres, h_spheres,
                          numProblems * numSpheres * sizeof(Sphere),
                          hipMemcpyHostToDevice));

  // Kernel launches
  int threadsPerBlock = 32;
  int blocksPerGridSpheres = numProblems;
  DetectSphereCollisions<<<blocksPerGridSpheres, threadsPerBlock>>>(
      d_spheres, numProblems, numSpheres, d_collisionMatrixSpheres);
  HANDLE_ERROR(hipDeviceSynchronize());

  // Copy results back to host
  HANDLE_ERROR(
      hipMemcpy(h_collisionMatrixSpheres, d_collisionMatrixSpheres,
                 numProblems * numSpheres * numSpheres * sizeof(CollisionData),
                 hipMemcpyDeviceToHost));

  // Free device memory
  hipFree(d_spheres);
  hipFree(d_collisionMatrixSpheres);
}
